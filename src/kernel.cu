#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "debug_util.cuh"
#include "geometry.cuh"
#include "bsdf.cuh"
#include "sampler.cuh"
#include "sky.cuh"
#include "postprocessing.cuh"
#include "raygen.cuh"
#include "traverse.cuh"
#include "light.cuh"

__device__ inline void LightShader(ConstBuffer& cbo, RayState& rayState, SceneMaterial sceneMaterial, TexObj skyTex)
{
	// check for termination and hit light
	if (rayState.hitLight == false) { return; }

	Float3 beta = rayState.beta;
	Float3 lightDir = rayState.dir;

	// Different light source type
	if (rayState.matType == MAT_SKY)
	{
		// env light
		//Float3 envLightColor = EnvLight(lightDir, cbo.sunDir, cbo.clockTime, rayState.isDiffuseRay);
		//Float3 envLightColor = Float3(0.8f);
		Float3 envLightColor = EnvLight2(lightDir, cbo.clockTime, rayState.isDiffuseRay, skyTex);
		rayState.L += envLightColor * beta;
	}
	else if (rayState.matType == EMISSIVE)
	{
		// local light
		SurfaceMaterial mat = sceneMaterial.materials[rayState.matId];
		Float3 L0 = mat.albedo;
		rayState.L += L0 * beta;
	}
}

__device__ inline void GlossyShader(ConstBuffer& cbo, RayState& rayState, SceneMaterial sceneMaterial, BlueNoiseRandGenerator& randGen, int loopIdx)
{
	// check for termination and hit light
	if (rayState.hitLight == true || rayState.isDiffuse == true) { return; }

	if (loopIdx == 0) { rayState.bounceLimit = 4; }

	if (rayState.matType == PERFECT_REFLECTION)
	{
		// mirror
		rayState.dir = normalize(rayState.dir - rayState.normal * dot(rayState.dir, rayState.normal) * 2.0);
		rayState.orig = rayState.pos + rayState.offset * rayState.normal;
	}
	else if (rayState.matType == PERFECT_FRESNEL_REFLECTION_REFRACTION)
	{
		// glass
		Float3 nextRayDir;
		float rayOffset = rayState.offset;
		float surfaceRand = randGen.Rand(4 + loopIdx * 6 + 0);
		PerfectReflectionRefraction(1.0, 1.33, rayState.isRayIntoSurface, rayState.normal, rayState.normalDotRayDir, surfaceRand, rayState.dir, nextRayDir, rayOffset);
		rayState.dir = nextRayDir;
		rayState.orig = rayState.pos + rayOffset * rayState.normal;
	}
}

__device__ inline void DiffuseShader(ConstBuffer& cbo, RayState& rayState, SceneMaterial sceneMaterial, SceneTextures textures, float* skyCdf, BlueNoiseRandGenerator& randGen, int loopIdx)
{
	// check for termination and hit light
	if (rayState.hitLight == true || rayState.isDiffuse == false) { return; }

	rayState.isDiffuseRay = true;

	// get mat
	SurfaceMaterial mat = sceneMaterial.materials[rayState.matId];

	Float3 albedo;
#if 0
	float uvScale = 60.0f;
	if (mat.useTex0)
	{
		float4 texColor = tex2D<float4>(textures.array[mat.texId0], rayState.uv.x * uvScale, rayState.uv.y * uvScale);
		albedo = Float3(texColor.x, texColor.y, texColor.z);
	}
	else
	{
		albedo = mat.albedo;
	}
#else
	albedo = mat.albedo;
#endif

	Float3 normal = rayState.normal;
#if 0
	if (mat.useTex1)
	{
		float4 texColor = tex2D<float4>(textures.array[mat.texId1], rayState.uv.x * uvScale, rayState.uv.y * uvScale);
		Float3 texNormal = Float3(texColor.x - 0.5, texColor.y - 0.5, texColor.z * 0.5);

		Float3 tangent = Float3(0, 1, 0);

		if (normal.y > 1.0f - 1e-3f)
			tangent = Float3(1, 0, 0);

		Float3 bitangent = cross(normal, tangent);
		tangent = cross(normal, bitangent);

		texNormal = normalize(tangent * texNormal.x + bitangent * texNormal.y + normal * texNormal.z);

		normal = texNormal;
		//normal = mixf(normal, texNormal, 0.0f);
	}
#endif

	Float3 rayDir = rayState.dir;

	// light sample
	float isDeltaLight = false;
	Float3 lightSampleDir;
	float lightSamplePdf;
	bool isLightSampled = SampleLight(cbo, rayState, sceneMaterial, lightSampleDir, lightSamplePdf, isDeltaLight, skyCdf, randGen, loopIdx);

	// surface sample
	Float3 surfSampleDir;

	Float3 surfaceBsdfOverPdf;
	Float3 surfaceSampleBsdf;
	float surfaceSamplePdf = 0;

	Float3 lightSampleSurfaceBsdfOverPdf;
	Float3 lightSampleSurfaceBsdf;
	float lightSampleSurfacePdf = 0;

	Float2 surfaceDiffuseRand2 = randGen.Rand2(4 + loopIdx * 6 + 0);
	if (rayState.matType == LAMBERTIAN_DIFFUSE)
	{
		LambertianSample(surfaceDiffuseRand2, surfSampleDir, normal);

		if (isDeltaLight == false)
		{
			surfaceBsdfOverPdf = LambertianBsdfOverPdf(albedo);
			surfaceSampleBsdf = LambertianBsdf(albedo);
			surfaceSamplePdf = LambertianPdf(surfSampleDir, normal);
		}

		if (isLightSampled == true)
		{
			lightSampleSurfaceBsdfOverPdf = LambertianBsdfOverPdf(albedo);
			lightSampleSurfaceBsdf = LambertianBsdf(albedo);
			lightSampleSurfacePdf = LambertianPdf(lightSampleDir, normal);
		}
	}
	else if (rayState.matType == MICROFACET_REFLECTION)
	{
		Float3 F0 = mat.F0;
		float alpha = mat.alpha;

		if (isDeltaLight == false)
			MacrofacetReflectionSample(surfaceDiffuseRand2, rayDir, surfSampleDir, normal, surfaceBsdfOverPdf, surfaceSampleBsdf, surfaceSamplePdf, F0, albedo, alpha);

		if (isLightSampled == true)
			MacrofacetReflection(lightSampleSurfaceBsdfOverPdf, lightSampleSurfaceBsdf, lightSampleSurfacePdf, normal, lightSampleDir, rayDir, F0, albedo, alpha);
	}

	// -------------------------------------- MIS balance heuristic ------------------------------------------
	float misRand = randGen.Rand(4 + loopIdx * 6 + 4);
	if (isLightSampled)
	{
		if (isDeltaLight)
		{
			// if a delta light (or say distant/directional light, typically sun light) is sampled,
			// no surface sample is needed since the weight for surface is zero
			rayState.beta *= lightSampleSurfaceBsdf;
			rayState.dir = lightSampleDir;
		}
		else
		{
			// The full equation for MIS is L = sum w_i * f_i / pdf_i
			// which in this case, two samples, one from surface bsdf distribution, one from light distribution
			//
			// L = w_surf * bsdf(dir_surf) / surfaceSamplePdf(dir_surf) + w_light * bsdf(dir_light) / surfaceSamplePdf(dir_light)
			// where w_surf = surfaceSamplePdf(dir_surf) / (surfaceSamplePdf(dir_surf) + lightSamplePdf)
			//       w_light = surfaceSamplePdf(dir_light) / (surfaceSamplePdf(dir_light) + lightSamplePdf)
			//
			// Then it'll become
			// L = bsdf(dir_surf) / (surfaceSamplePdf(dir_surf) + lightSamplePdf) +
			//     bsdf(dir_light) / (surfaceSamplePdf(dir_light) + lightSamplePdf)
			//
			// My algorithm takes bsdf as value and misWeight*pdf as weight,
			// using the weights to choose either sample light or surface.
			// It achieves single ray sample per surface shader with no bias to MIS balance heuristic algorithm
			float lightSampleWeight =  1.0f / (lightSamplePdf + lightSampleSurfacePdf);
			float surfaceSampleWeight = 1.0f / (surfaceSamplePdf + lightSamplePdf);

			float chooseSurfaceFactor = surfaceSampleWeight / (lightSampleWeight + surfaceSampleWeight);

			if (misRand < chooseSurfaceFactor)
			{
				// choose surface scatter sample
				rayState.beta *= min3f(surfaceSampleBsdf * surfaceSampleWeight / chooseSurfaceFactor, Float3(1.0f));
				rayState.dir = surfSampleDir;
			}
			else
			{
				// choose light sample
				rayState.beta *= min3f(lightSampleSurfaceBsdf * lightSampleWeight / (1.0f - chooseSurfaceFactor), Float3(1.0f));
				rayState.dir = lightSampleDir;
			}
		}
	}
	else
	{
		// if no light sample condition is met, sample surface only, which is the vanila case
		rayState.beta *= surfaceBsdfOverPdf;
		rayState.dir = surfSampleDir;
	}

	rayState.orig = rayState.pos + rayState.offset * normal;
}

__global__ void PathTrace(ConstBuffer cbo, SceneGeometry sceneGeometry, SceneMaterial sceneMaterial, BlueNoiseRandGenerator randGen, SurfObj colorBuffer, SceneTextures textures, TexObj skyTex, float* skyCdf)
{
	// index
	Int2 idx(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	int i = gridDim.x * blockDim.x * idx.y + idx.x;

	int sampleNum = 1;
	float encodedNormal;
	Float3 outColor = 0;

	#pragma unroll
	for (int sampleIdx = 0; sampleIdx < sampleNum; ++sampleIdx)
	{
		// init ray state
		RayState rayState;
		rayState.i            = i;
		rayState.L            = 0.0;
		rayState.beta         = 1.0;
		rayState.idx          = idx;
		rayState.isDiffuseRay = false;
		rayState.hitLight     = false;
		rayState.bounceLimit  = 2;

		// setup rand gen
		randGen.idx = idx;
		randGen.sampleIdx = cbo.frameNum * sampleNum + sampleIdx;

		// generate ray
		GenerateRay(rayState.orig, rayState.dir, cbo.camera, idx, randGen.Rand2(0), randGen.Rand2(2));
		RaySceneIntersect(cbo, sceneMaterial, sceneGeometry, rayState);

		// encode normal
		if (sampleIdx == 0) { encodedNormal = EncodeNormal_R11_G10_B11(rayState.normal); }

		// main loop
		int loopIdx = 0;
		for (; loopIdx < rayState.bounceLimit; ++loopIdx)
		{
			GlossyShader(cbo, rayState, sceneMaterial, randGen, loopIdx);
			DiffuseShader(cbo, rayState, sceneMaterial, textures, skyCdf, randGen, loopIdx);
			RaySceneIntersect(cbo, sceneMaterial, sceneGeometry, rayState);
		}

		GlossyShader(cbo, rayState, sceneMaterial, randGen, loopIdx);
		LightShader(cbo, rayState, sceneMaterial, skyTex);

		outColor += rayState.L;
	}

	// write to buffer
	Store2D(Float4(outColor / (float)sampleNum, encodedNormal), colorBuffer, idx);
}

void RayTracer::UpdateFrame()
{
	// timer
	timer.update();
	deltaTime = timer.getDeltaTime();
	clockTime = timer.getTime();
	cbo.clockTime     = clockTime;

	// sun dir
    const Float3 axis = normalize(Float3(1.0f, 0.0f, -0.4f));
	const float angle = fmodf(clockTime * TWO_PI / 100, TWO_PI);
	sunDir     = rotate3f(axis, angle, Float3(0.0, -1.0, 0.0)).normalized();
	cbo.sunDir        = sunDir;

	// frame number
	static int framen = 1;
	cbo.frameNum      = framen++;

	// prepare for lens flare
	Camera& camera = cbo.camera;
	Mat3 invCamMat(camera.left, camera.up, camera.dir); // build view matrix
	invCamMat.transpose(); // orthogonal matrix, inverse is transpose
	Float3 sunPosViewSpace = sunDir * invCamMat; // transform sun dir to view space
	sunPos = sunPosViewSpace.xy; // get xy
	sunPos /= sunPosViewSpace.z; // get the x and y when z is 1
	sunPos /= camera.tanHalfFov; // [-1, 1]
	sunPos = Float2(0.5) - sunPos * Float2(0.5); // [0, 1]
}

void RayTracer::draw(SurfObj* renderTarget)
{
	// update frame
	UpdateFrame();

	// dimensions
	Int2 bufferDim(renderWidth, renderHeight);
	Int2 outputDim(screenWidth, screenHeight);

	// ---------------- init histogram ----------------
	GpuErrorCheck(hipMemsetAsync(d_histogram, 0, 64 * sizeof(uint), streams[0]));

	// ---------------- sky ----------------
	Sky<<<dim3(8, 2, 1), dim3(8, 8, 1), 0, streams[0]>>>(skyBuffer, skyCdf, Int2(64, 16), sunDir);
	Scan<<<1, dim3(512, 1, 1), 1024 * sizeof(float), streams[0]>>>(skyCdf, 1024);

	// ---------------- path tracing ----------------------
	PathTrace<<<gridDim, blockDim, 0, streams[0]>>>(cbo, d_sceneGeometry, d_sceneMaterial, d_randGen, colorBufferA, sceneTextures, skyTex, skyCdf);

	DEBUG_CUDA();

	// ---------------- post processing -------------------
	if (cbo.frameNum != 1)
	{
		TemporalFilter <<<gridDim, blockDim, 0, streams[0]>>>(colorBufferA, colorBufferB, bufferDim);
	}

	// Denoise
	AtousFilter<<<dim3(divRoundUp(renderWidth, 28), divRoundUp(renderHeight, 28), 1), dim3(32, 32, 1), 0, streams[0]>>>(colorBufferA, colorBufferB, bufferDim);

	// Histogram
	DownScale4_fp32_fp16<<<gridDim, blockDim, 0, streams[0]>>>(colorBufferA, colorBuffer4, bufferDim);
	DownScale4_fp16_fp16<<<gridDim4, blockDim, 0, streams[0]>>>(colorBuffer4, colorBuffer16, bufferSize4);
	DownScale4_fp16_fp16<<<gridDim16, blockDim, 0, streams[0]>>>(colorBuffer16, colorBuffer64, bufferSize16);

	Histogram2<<<1, dim3(bufferSize64.x, bufferSize64.y, 1), 0, streams[0]>>>(/*out*/d_histogram, /*in*/colorBuffer64 , bufferSize64);

	// Exposure
	AutoExposure<<<1, 1, 0, streams[0]>>>(/*out*/d_exposure, /*in*/d_histogram, (float)(bufferSize64.x * bufferSize64.y), deltaTime);

	// Bloom
	BloomGuassian<<<dim3(divRoundUp(bufferSize4.x, 12), divRoundUp(bufferSize4.y, 12), 1), dim3(16, 16, 1), 0, streams[0]>>>(bloomBuffer4, colorBuffer4, bufferSize4, d_exposure);
	BloomGuassian<<<dim3(divRoundUp(bufferSize16.x, 12), divRoundUp(bufferSize16.y, 12), 1), dim3(16, 16, 1), 0, streams[0]>>>(bloomBuffer16, colorBuffer16, bufferSize16, d_exposure);

	Bloom<<<gridDim, blockDim, 0, streams[0]>>>(colorBufferA, bloomBuffer4, bloomBuffer16, bufferDim, bufferSize4, bufferSize16);

	// Lens flare
	if (sunPos.x > 0 && sunPos.x < 1 && sunPos.y > 0 && sunPos.y < 1 && sunDir.y > -0.0 && dot(sunDir, cbo.camera.dir) > 0)
	{
		sunPos -= Float2(0.5);
		sunPos.x *= (float)renderWidth / (float)renderHeight;
		LensFlare<<<gridDim, blockDim, 0, streams[0]>>>(sunPos, colorBufferA, bufferDim);
	}

	// Tone mapping
	ToneMapping<<<gridDim, blockDim, 0, streams[0]>>>(colorBufferA , bufferDim , d_exposure);

	// Scale to final output
	FilterScale<<<scaleGridDim, scaleBlockDim, 0, streams[0]>>>(/*out*/renderTarget, /*in*/colorBufferA, outputDim, bufferDim);
}