#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "debug_util.cuh"
#include "geometry.cuh"
#include "bsdf.cuh"
#include "morton.cuh"
#include "sampler.cuh"
#include "denoise.cuh"
#include "sky.cuh"
#include "postprocessing.cuh"
#include "raygen.cuh"
#include "traverse.cuh"
#include "hash.cuh"

__device__ inline void LightShader(ConstBuffer& cbo, RayState& rayState, SceneMaterial sceneMaterial)
{
	// check for termination and hit light
	if (rayState.terminated == true || rayState.hitLight == false) { return; }

	// ray is terminated
	rayState.terminated = true;

	// Different light source type
	if (rayState.matType == MAT_SKY)
	{
		// env light
		Float3 envLightColor = EnvLight(rayState.dir, cbo.sunDir);
		rayState.L += envLightColor * rayState.beta;
	}
	else if (rayState.matType == EMISSIVE)
	{
		// local light
		SurfaceMaterial mat = sceneMaterial.materials[rayState.matId];
		Float3 L0 = mat.albedo;
		rayState.L += L0 * rayState.beta;
	}
}

__device__ inline void GlossyShader(ConstBuffer& cbo, RayState& rayState, SceneMaterial sceneMaterial)
{
	// check for termination and hit light
	if (rayState.terminated == true || rayState.hitLight == true || rayState.isDiffuse == true) { return; }

	if (rayState.matType == PERFECT_REFLECTION)
	{
		// mirror
		rayState.dir = normalize(rayState.dir - rayState.normal * dot(rayState.dir, rayState.normal) * 2.0);
		rayState.orig = rayState.pos + rayState.offset * rayState.normal;
	}
	else if (rayState.matType == PERFECT_FRESNEL_REFLECTION_REFRACTION)
	{
		// glass
		Float3 nextRayDir;
		float rayOffset = rayState.offset;
		PerfectReflectionRefraction(1.0, 1.33, rayState.isRayIntoSurface, rayState.normal, rayState.normalDotRayDir, rd(&rayState.rdState[2]), rayState.dir, nextRayDir, rayOffset);
		rayState.dir = nextRayDir;
		rayState.orig = rayState.pos + rayOffset * rayState.normal;
	}
}

__device__ inline void DiffuseShader(ConstBuffer& cbo, RayState& rayState, SceneMaterial sceneMaterial)
{
	// check for termination and hit light
	if (rayState.terminated == true || rayState.hitLight == true || rayState.isDiffuse == false) { return; }

	// get mat
	SurfaceMaterial mat = sceneMaterial.materials[rayState.matId];

	if (rayState.matType == LAMBERTIAN_DIFFUSE)
	{
		// surface sample
		LambertianSample(rd2(&rayState.rdState[0], &rayState.rdState[1]), rayState.dir, rayState.normal);
		Float3 surfaceSamplebsdfOverPdf = LambertianBsdfOverPdf(mat.albedo);
		float surfaceSamplePdf = LambertianPdf(rayState.dir, rayState.normal);

		// light sample
		/// @todo

		// update ray
		rayState.orig = rayState.pos + rayState.offset * rayState.normal;

		// update beta
		rayState.beta *= surfaceSamplebsdfOverPdf;
	}
	else if (rayState.matType == MICROFACET_REFLECTION)
	{
		const Float3 F0(0.56, 0.57, 0.58);
		const float alpha = 0.05;

		Float3 nextRayDir;
		MacrofacetReflection(rd(&rayState.rdState[0]), rd(&rayState.rdState[1]), rayState.dir, nextRayDir, rayState.normal, rayState.beta, F0, alpha);
		rayState.dir = nextRayDir;

		// light sample
		/// @todo

		// update ray
		rayState.orig = rayState.pos + rayState.offset * rayState.normal;
	}
}

__device__ inline void UpdateMaterial(RayState& rayState, SceneMaterial sceneMaterial)
{
	if (rayState.objectIdx == 998)
	{
		rayState.matType = PERFECT_REFLECTION;
	}
	else
	{
		rayState.matId = sceneMaterial.materialsIdx[rayState.objectIdx];
		SurfaceMaterial mat = sceneMaterial.materials[rayState.matId];
		rayState.matType = (rayState.hit == false) ? MAT_SKY : mat.type;
	}
	rayState.hitLight = (rayState.matType == MAT_SKY) || (rayState.matType == EMISSIVE);
	rayState.isDiffuse = (rayState.matType == LAMBERTIAN_DIFFUSE) || (rayState.matType == MICROFACET_REFLECTION);
}

__global__ void PathTrace(ConstBuffer cbo, SceneGeometry sceneGeometry, SceneMaterial sceneMaterial, RandInitVec* randInitVec, SurfObj colorBuffer, SurfObj normalBuffer, SurfObj positionBuffer)
{
	// index
	Int2 idx(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	int i = gridDim.x * blockDim.x * idx.y + idx.x;

	// init ray state
	RayState rayState;
	rayState.i          = i;
	rayState.L          = 0.0;
	rayState.beta       = 1.0;
	rayState.idx        = idx;
	rayState.bounce     = 0;
	rayState.terminated = false;
	rayState.distance   = 0;

	// init rand state
	const uint seed = (idx.x << 16) ^ (idx.y);
	for (int k = 0; k < 3; ++k) { hiprand_init(randInitVec[k], CURAND_2POW32 * 0.5f, WangHash(seed) + cbo.frameNum, &rayState.rdState[k]); }

	// generate ray
	GenerateRay(rayState.orig, rayState.dir, cbo.camera, idx, rd2(&rayState.rdState[0], &rayState.rdState[1]));

	// scene traverse
	rayState.hit = RaySceneIntersect(Ray(rayState.orig, rayState.dir), sceneGeometry, rayState.pos, rayState.normal, rayState.objectIdx, rayState.offset, rayState.distance, rayState.isRayIntoSurface, rayState.normalDotRayDir);

	// store normal, pos
	Store2D(Float4(rayState.normal, 1.0), normalBuffer, idx);
	Store2D(Float4(rayState.pos, 1.0), positionBuffer, idx);

	// update mat id
	UpdateMaterial(rayState, sceneMaterial);

	for (int bounce = 0; bounce < 4; ++bounce)
	{
		LightShader(cbo, rayState, sceneMaterial);
		GlossyShader(cbo, rayState, sceneMaterial);
		DiffuseShader(cbo, rayState, sceneMaterial);

		if (bounce != 3)
		{
			rayState.hit = RaySceneIntersect(Ray(rayState.orig, rayState.dir), sceneGeometry, rayState.pos, rayState.normal, rayState.objectIdx, rayState.offset, rayState.distance, rayState.isRayIntoSurface, rayState.normalDotRayDir);
			UpdateMaterial(rayState, sceneMaterial);
		}
	}

	// write to buffer
	Store2D(Float4(rayState.L, 1.0), colorBuffer, idx);
}

void RayTracer::draw(SurfObj* renderTarget)
{
	// ---------------- frame update ------------------
	timer.update();
	float clockTime   = timer.getTime();

	const Float3 axis = normalize(Float3(0.0, 0.0, 1.0));
	const float angle = fmodf(clockTime * TWO_PI / 10, TWO_PI);
	Float3 sunDir     = rotate3f(axis, angle, Float3(0.0, 1.0, 2.5)).normalized();

	cbo.sunDir        = sunDir;
	cbo.frameNum      = cbo.frameNum + 1;

	// ----- scene -----
	// sphere
	Float3 spherePos = Float3(sinf(clockTime * TWO_PI / 5) * 0.01f, terrain.getHeightAt(0.0f) + 0.005f, 0.0f);
	cameraFocusPos        = Float3(0.0f, terrain.getHeightAt(0.0f) + 0.005f, 0.0f);
	spheres[0]            = Sphere(spherePos, 0.005f);
	sphereLights[0]           = spheres[0];

	GpuErrorCheck(hipMemcpyAsync(d_spheres          , spheres      , numSpheres *      sizeof(Float4)         , hipMemcpyHostToDevice, streams[2]));
	GpuErrorCheck(hipMemcpyAsync(d_sphereLights     , sphereLights , numSphereLights * sizeof(Float4)         , hipMemcpyHostToDevice, streams[2]));

	d_sceneGeometry.numSpheres      = numSpheres;
	d_sceneGeometry.spheres         = d_spheres;

	d_sceneMaterial.numSphereLights = numSphereLights;
	d_sceneMaterial.sphereLights    = d_sphereLights;

	// camera
	// Camera& camera = cbo.camera;
	// Float3 cameraLookAtPoint = cameraFocusPos + Float3(0.0f, 0.01f, 0.0f);
	// camera.pos               = cameraFocusPos + rotate3f(Float3(0, 1, 0), fmodf(clockTime * TWO_PI / 60, TWO_PI), Float3(0.0f, 0.0f, -0.1f)) + Float3(0, abs(sinf(clockTime * TWO_PI / 60)) * 0.05f, 0);
	// camera.dir               = normalize(cameraLookAtPoint - camera.pos.xyz);
	// camera.left              = cross(Float3(0, 1, 0), camera.dir.xyz);

	Int2 bufferDim(renderWidth, renderHeight);
	Int2 outputDim(screenWidth, screenHeight);

	// ------------------ Ray Gen -------------------
	PathTrace<<<gridDim, blockDim, 0, streams[0]>>>(cbo, d_sceneGeometry, d_sceneMaterial, d_randInitVec, colorBufferA, normalBuffer, positionBuffer);

	// ---------------- post processing ----------------
	ToneMapping<<<gridDim, blockDim, 0, streams[0]>>>(/*io*/colorBufferA , bufferDim , /*exposure*/1.0);
	Denoise    <<<gridDim, blockDim, 0, streams[0]>>>(/*io*/colorBufferA , /*in*/normalBuffer , /*in*/positionBuffer, bufferDim, cbDenoise);

	if (cbo.frameNum == 1) { BufferCopy<<<gridDim, blockDim, 0, streams[0]>>>(/*out*/colorBufferB , /*in*/colorBufferA , bufferDim); }
	else                   { TAA       <<<gridDim, blockDim, 0, streams[0]>>>(/*io*/colorBufferB  , /*in*/colorBufferA , bufferDim); }

	FilterScale<<<scaleGridDim, scaleBlockDim, 0, streams[0]>>>(/*out*/renderTarget, /*in*/colorBufferB, outputDim, bufferDim);
}