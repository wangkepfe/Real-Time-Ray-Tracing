#include "hip/hip_runtime.h"

#include "kernel.cuh"
#include "fileUtils.cuh"
#include "blueNoiseRandGenData.h"
#include "hip/hip_fp16.h"

extern GlobalSettings* g_settings;

template<typename T>
__global__ void InitBuffer(T val, SurfObj buffer, Int2 bufferSize)
{
	Int2 idx;
	idx.x = blockIdx.x * blockDim.x + threadIdx.x;
	idx.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx.x >= bufferSize.x || idx.y >= bufferSize.y) return;

	surf2Dwrite(val, buffer, idx.x, idx.y, hipBoundaryModeClamp);
}

void RayTracer::init(hipStream_t* cudaStreams)
{
	maxRenderWidth = g_settings->maxWidth;
	maxRenderHeight = g_settings->maxHeight;

	if (g_settings->useDynamicResolution)
	{
		renderWidth = maxRenderWidth;
		renderHeight = maxRenderHeight;
	}
	else
	{
		renderWidth = screenWidth;
		renderHeight = screenHeight;
	}

	uint i;

	// set streams
	streams = cudaStreams;

	// init cuda
	gpuDeviceInit(0);

	{
		// load triangles
		std::vector<Triangle> h_triangles;

		//const char* filename = "resources/models/test.dae";
		//LoadScene(filename, h_triangles);

		std::string fileName = g_settings->inputMeshFileName;
		std::ifstream infile (fileName, std::ifstream::binary);
		if (infile.good())
		{
			size_t currentSize = sizeof(uint);
			char* pTriCount = new char[currentSize];
			infile.read(pTriCount, currentSize);
			triCount = *reinterpret_cast<uint*>(pTriCount);

			currentSize = sizeof(Triangle) * triCount;
			char* pTrianglesRaw = new char[currentSize];
			infile.read(pTrianglesRaw, currentSize);
			Triangle* pTriangles = reinterpret_cast<Triangle*>(pTrianglesRaw);
			h_triangles.assign(pTriangles, pTriangles + triCount);

			infile.close();
			std::cout << "Successfully read scene data from \"" << fileName << "\"!\n";
		} else {
			std::cout << "Error: Failed to read scene data from \"" << fileName << "\".\n";
		}

		//triCount = static_cast<uint>(h_triangles.size());
		// pad the tricount to a multiply of BatchSize
		triCountPadded = triCount;
		if (triCountPadded % BatchSize != 0)
		{
			triCountPadded += (BatchSize - triCountPadded % BatchSize);

			h_triangles.resize(triCountPadded);

			// repeat the last triangle for a few times
			for (int i = triCount; i < triCountPadded; ++i)
			{
				h_triangles[i] = h_triangles[triCount - 1];
			}
		}

		GpuErrorCheck(hipMalloc((void**)& constTriangles, triCountPadded * sizeof(Triangle)));
		GpuErrorCheck(hipMemcpy(constTriangles, h_triangles.data(), triCountPadded * sizeof(Triangle), hipMemcpyHostToDevice));

		// batch count
		batchCount = triCountPadded / BatchSize;

		GpuErrorCheck(hipMalloc((void**)& batchCountArray, 1 * sizeof(uint)));
		GpuErrorCheck(hipMemcpy(batchCountArray, &batchCount, 1 * sizeof(uint), hipMemcpyHostToDevice));

		// triangle batch count array
		std::vector<uint> h_triCountArray(batchCount, BatchSize);
		h_triCountArray[batchCount - 1] = triCount - (triCountPadded - BatchSize);

		GpuErrorCheck(hipMalloc((void**)& triCountArray, batchCount * sizeof(uint)));
		GpuErrorCheck(hipMemcpy(triCountArray, h_triCountArray.data(), batchCount * sizeof(uint), hipMemcpyHostToDevice));

		// pad the batch count to a multiply of KernalBatchSize
		batchCountPadded = batchCount;
		if (batchCountPadded % KernalBatchSize != 0)
		{
			batchCountPadded += (KernalBatchSize - batchCountPadded % KernalBatchSize);
		}
	}

	// -------------------------------- bvh ---------------------------------------
	// triangle
	GpuErrorCheck(hipMalloc((void**)& triangles, triCountPadded * sizeof(Triangle)));

	// aabb
	GpuErrorCheck(hipMalloc((void**)& aabbs, triCountPadded * sizeof(AABB)));

	// morton code
	GpuErrorCheck(hipMalloc((void**)& morton, triCountPadded * sizeof(uint)));
	GpuErrorCheck(hipMemset(morton, UINT_MAX, triCountPadded * sizeof(uint))); // init morton code to UINT_MAX

	// reorder idx
	GpuErrorCheck(hipMalloc((void**)& reorderIdx, triCountPadded * sizeof(uint)));

	// bvh nodes
	GpuErrorCheck(hipMalloc((void**)& bvhNodes, triCountPadded * sizeof(BVHNode)));

	//------------------------------------ tlas -------------------------------------------

	// aabb
	GpuErrorCheck(hipMalloc((void**)& tlasAabbs, BatchSize * sizeof(AABB)));

	// morton code
	GpuErrorCheck(hipMalloc((void**)& tlasMorton, BatchSize * sizeof(uint)));
	GpuErrorCheck(hipMemset(tlasMorton, UINT_MAX, BatchSize * sizeof(uint))); // init morton code to UINT_MAX

	// reorder idx
	GpuErrorCheck(hipMalloc((void**)& tlasReorderIdx, BatchSize * sizeof(uint)));

	// bvh nodes
	GpuErrorCheck(hipMalloc((void**)& tlasBvhNodes, BatchSize * sizeof(BVHNode)));

	//-------------------------------------------------------------------------------

	// AABB
	int numAabbs = 2;
	sceneAabbs = new AABB[numAabbs];
	i = 0;
	sceneAabbs[i++] = AABB({0.0f, 0.0f, 0.0f}, {0.0f, 0.0f, 0.0f});
	sceneAabbs[i++] = AABB({0.0f, 0.0f, 0.0f}, {0.0f, 0.0f, 0.0f});

	// sphere
	numSpheres = 2;
	spheres    = new Sphere[numSpheres];
	i = 0;
	spheres[i++] = Sphere({0.0f, 1.0f, 4.0f}, 1.0f);
	spheres[i++] = Sphere({0.0f, 1.0f, -4.0f}, 1.0f);

	// surface materials
	const int numMaterials     = 10;
	SurfaceMaterial* materials = new SurfaceMaterial[numMaterials];
	i = 0;
	materials[i].type          = EMISSIVE;
	materials[i].albedo        = Float3(0.1f, 0.2f, 0.9f);
	++i;
	materials[i].type          = PERFECT_FRESNEL_REFLECTION_REFRACTION;
	++i;
	materials[i].type          = EMISSIVE;
	materials[i].albedo        = Float3(0.9f, 0.2f, 0.1f);
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].albedo        = Float3(0.9f);
	++i;
	materials[i].type          = MICROFACET_REFLECTION;
	materials[i].albedo        = Float3(0.9f);
	materials[i].F0            = Float3(0.56f, 0.57f, 0.58f);
	materials[i].alpha         = 0.05f;
	++i;
	materials[i].type          = MICROFACET_REFLECTION;
	materials[i].albedo        = Float3(0.9f);
	materials[i].F0            = Float3(0.56f, 0.57f, 0.58f);
	materials[i].alpha         = 0.01f;
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].useTex0       = true;
	materials[i].texId0        = 0;
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].albedo        = Float3(0.9f, 0.2f, 0.1f);
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].albedo        = Float3(0.2f, 0.9f, 0.1f);
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].albedo        = Float3(0.1f, 0.2f, 0.9f);

	// number of objects
	const int numObjects = triCount + numSpheres;

	// material index
	int* materialsIdx = new int[numObjects];
	for (i = 0; i < triCount; ++i)
	{
		materialsIdx[i] = 4;
	}
	materialsIdx[i++] = 0;
	materialsIdx[i++] = 2;

	// light source
	numSphereLights = 2;
	sphereLights = new Sphere[numSphereLights];
	for (i = 0; i < numSphereLights; ++i)
	{
		sphereLights[i] = spheres[i];
	}

	// constant buffer
	cbo.frameNum = 0;
	cbo.bvhDebugLevel = -1;
	cbo.bvhBatchSize = BatchSize;

	// launch param
	blockDim = dim3(8, 8, 1);
	gridDim = dim3(divRoundUp(renderWidth, blockDim.x), divRoundUp(renderHeight, blockDim.y), 1);

	scaleBlockDim = dim3(8, 8, 1);
	scaleGridDim = dim3(divRoundUp(screenWidth, scaleBlockDim.x), divRoundUp(screenHeight, scaleBlockDim.y), 1);

	// ------------------------ surface/texture object ---------------------------
	hipChannelFormatDesc format_color_RGB16_mask_A16 = cudaCreateChannelDescHalf4();
	hipChannelFormatDesc format_normal_R11_G10_B11_depth_R32 = hipCreateChannelDesc<float2>();
	hipChannelFormatDesc format_motionVector_UV16 = cudaCreateChannelDescHalf2();
	hipChannelFormatDesc format_sampleCount_R8 = hipCreateChannelDesc<uchar1>();
	hipChannelFormatDesc format_R16 = cudaCreateChannelDescHalf1();
	hipChannelFormatDesc format_RGBA_FLOAT16 = cudaCreateChannelDescHalf4();

	// resource desription
	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;

	// texture description
	hipTextureDesc texDesc  = {};
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.addressMode[1]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	// array A: main render buffer
	GpuErrorCheck(hipMallocArray(&colorBufferArrayA, &format_color_RGB16_mask_A16, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArrayA;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBufferA, &resDesc));

	// array B: TAA buffer
	GpuErrorCheck(hipMallocArray(&colorBufferArrayB, &format_color_RGB16_mask_A16, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArrayB;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBufferB, &resDesc));

	// normal depth
	GpuErrorCheck(hipMallocArray(&normalDepthBufferArrayA, &format_normal_R11_G10_B11_depth_R32, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = normalDepthBufferArrayA;
	GpuErrorCheck(hipCreateSurfaceObject(&normalDepthBufferA, &resDesc));

	GpuErrorCheck(hipMallocArray(&normalDepthBufferArrayB, &format_normal_R11_G10_B11_depth_R32, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = normalDepthBufferArrayB;
	GpuErrorCheck(hipCreateSurfaceObject(&normalDepthBufferB, &resDesc));

	// motion vector buffer
	GpuErrorCheck(hipMallocArray(&motionVectorBufferArray, &format_motionVector_UV16, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = motionVectorBufferArray;
	GpuErrorCheck(hipCreateSurfaceObject(&motionVectorBuffer, &resDesc));

	// sample count buffer
	GpuErrorCheck(hipMallocArray(&sampleCountBufferArray, &format_sampleCount_R8, gridDim.x, gridDim.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = sampleCountBufferArray;
	GpuErrorCheck(hipCreateSurfaceObject(&sampleCountBuffer, &resDesc));

	InitBuffer<<<dim3(divRoundUp(gridDim.x, 8), divRoundUp(gridDim.y, 8), 1), dim3(8, 8, 1)>>> (make_uchar1(1), sampleCountBuffer, Int2(gridDim.x, gridDim.y));

	// noise level uffer
	GpuErrorCheck(hipMallocArray(&noiseLevelBufferArray, &format_R16, gridDim.x, gridDim.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = noiseLevelBufferArray;
	GpuErrorCheck(hipCreateSurfaceObject(&noiseLevelBuffer, &resDesc));

	InitBuffer<<<dim3(divRoundUp(gridDim.x, 8), divRoundUp(gridDim.y, 8), 1), dim3(8, 8, 1)>>> (make_ushort1(0), noiseLevelBuffer, Int2(gridDim.x, gridDim.y));

	// color buffer 1/4 size
	bufferSize4 = UInt2(divRoundUp(renderWidth, 4u), divRoundUp(renderHeight, 4u));
	gridDim4 = dim3(divRoundUp(bufferSize4.x, blockDim.x), divRoundUp(bufferSize4.y, blockDim.y), 1);
	GpuErrorCheck(hipMallocArray(&colorBufferArray4, &format_color_RGB16_mask_A16, bufferSize4.x, bufferSize4.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArray4;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBuffer4, &resDesc));

	// bloom buffer 1/4 size
	GpuErrorCheck(hipMallocArray(&bloomBufferArray4, &format_color_RGB16_mask_A16, bufferSize4.x, bufferSize4.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = bloomBufferArray4;
	GpuErrorCheck(hipCreateSurfaceObject(&bloomBuffer4, &resDesc));

	// color buffer 1/16 size
	bufferSize16 = UInt2(divRoundUp(bufferSize4.x, 4u), divRoundUp(bufferSize4.y, 4u));
	gridDim16 = dim3(divRoundUp(bufferSize16.x, blockDim.x), divRoundUp(bufferSize16.y, blockDim.y), 1);
	GpuErrorCheck(hipMallocArray(&colorBufferArray16, &format_color_RGB16_mask_A16, bufferSize16.x, bufferSize16.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArray16;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBuffer16, &resDesc));

	// bloom buffer 1/16 size
	GpuErrorCheck(hipMallocArray(&bloomBufferArray16, &format_color_RGB16_mask_A16, bufferSize16.x, bufferSize16.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = bloomBufferArray16;
	GpuErrorCheck(hipCreateSurfaceObject(&bloomBuffer16, &resDesc));

	// color buffer 1/64 size
	bufferSize64 = UInt2(divRoundUp(bufferSize16.x, 4u), divRoundUp(bufferSize16.y, 4u));
	gridDim64 = dim3(divRoundUp(bufferSize64.x, blockDim.x), divRoundUp(bufferSize64.y, blockDim.y), 1);
	GpuErrorCheck(hipMallocArray(&colorBufferArray64, &format_color_RGB16_mask_A16, bufferSize64.x, bufferSize64.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArray64;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBuffer64, &resDesc));

	// output
	GpuErrorCheck(hipMallocArray(&colorBufferArrayC, &format_color_RGB16_mask_A16, screenWidth, screenHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArrayC;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBufferC, &resDesc));

	//
	GpuErrorCheck(hipMallocArray(&bsdfOverPdfBufferArray, &format_RGBA_FLOAT16, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = bsdfOverPdfBufferArray;
	GpuErrorCheck(hipCreateSurfaceObject(&bsdfOverPdfBuffer, &resDesc));

	// ----------------------- sky buffer ------------------------
	GpuErrorCheck(hipMallocArray(&skyArray, &format_color_RGB16_mask_A16, skyWidth, skyHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = skyArray;
	GpuErrorCheck(hipCreateSurfaceObject(&skyBuffer, &resDesc));
	//GpuErrorCheck(hipCreateTextureObject(&skyTex, &resDesc, &texDesc, NULL));

	GpuErrorCheck(hipMalloc((void**)&skyCdf, skySize * sizeof(float)));
	GpuErrorCheck(hipMemset(skyCdf, 0, skySize * sizeof(float)));

	// ----------------------- GPU buffers -----------------------
	// exposure
	GpuErrorCheck(hipMalloc((void**)& d_exposure, 4 * sizeof(float)));
	float initExposureLum[4] = { 1.0f, 1.0f, 1.0f, 1.0f }; // (exposureValue, historyAverageLuminance, historyBrightThresholdLuminance, unused)
	GpuErrorCheck(hipMemcpy(d_exposure, initExposureLum, 4 * sizeof(float), hipMemcpyHostToDevice));

	// histogram
	GpuErrorCheck(hipMalloc((void**)& d_histogram, 64 * sizeof(uint)));

	// for debug
	GpuErrorCheck(hipMalloc((void**)& dumpFrameBuffer, screenWidth * screenHeight * sizeof(uchar4)));
	GpuErrorCheck(hipMemset(dumpFrameBuffer, 0, screenWidth * screenHeight * sizeof(uchar4)));

	// scene
	GpuErrorCheck(hipMalloc((void**)& d_spheres          , numSpheres *       sizeof(Sphere)));
	GpuErrorCheck(hipMalloc((void**)& d_sceneAabbs       , numAabbs *         sizeof(AABB)));
	GpuErrorCheck(hipMalloc((void**)& d_materialsIdx     , numObjects *       sizeof(int)));
	GpuErrorCheck(hipMalloc((void**)& d_surfaceMaterials , numMaterials *     sizeof(SurfaceMaterial)));
	GpuErrorCheck(hipMalloc((void**)& d_sphereLights     , numSphereLights *  sizeof(Float4)));

	GpuErrorCheck(hipMemcpy(d_spheres          , spheres      , numSpheres *      sizeof(Float4)         , hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(d_surfaceMaterials , materials    , numMaterials *    sizeof(SurfaceMaterial), hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(d_sceneAabbs       , sceneAabbs   , numAabbs *        sizeof(AABB)           , hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(d_materialsIdx     , materialsIdx , numObjects *      sizeof(int)            , hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(d_sphereLights     , sphereLights , numSphereLights * sizeof(Float4)         , hipMemcpyHostToDevice));

	// setup scene
	d_sceneGeometry.numSpheres      = numSpheres;
	d_sceneGeometry.spheres         = d_spheres;

	d_sceneGeometry.numAabbs        = numAabbs;
	d_sceneGeometry.aabbs           = d_sceneAabbs;

	d_sceneMaterial.numSphereLights = numSphereLights;
	d_sceneMaterial.sphereLights    = d_sphereLights;

	d_sceneMaterial.materials       = d_surfaceMaterials;
	d_sceneMaterial.materialsIdx    = d_materialsIdx;
	d_sceneMaterial.numMaterials    = numMaterials;

	d_sceneGeometry.triangles       = triangles;
	d_sceneGeometry.bvhNodes        = bvhNodes;
	d_sceneGeometry.tlasBvhNodes    = tlasBvhNodes;
	d_sceneGeometry.numTriangles    = triCount;

	delete[] materials;
	delete[] materialsIdx;

	// cuda random
	h_randGen.init();
	d_randGen = h_randGen;

	// camera
	CameraSetup(cbo.camera);

	// textures
	texArrayUv = LoadTextureRgba8(g_settings->inputTextureFileNames[0].c_str(), sceneTextures.uv);
	//texArraySandAlbedo = LoadTextureRgb8("resources/textures/sand.png", sceneTextures.sandAlbedo);
	//texArraySandNormal = LoadTextureRgb8("resources/textures/sand_n.png", sceneTextures.sandNormal);

	// timer init
	timer.init();

	// set render dim to screen dim
	renderWidth = screenWidth;
	renderHeight = screenHeight;
}

void RayTracer::CameraSetup(Camera& camera)
{
	//cameraFocusPos = Float3(0, 1.0f, 0);
	//camera.pos = cameraFocusPos + Float3(7.3f, 2.0f, -6.9f);
	camera.pos = Float3(4.3f, 1.4f, -3.9f);

	//Float3 cameraLookAtPoint = cameraFocusPos;
	//Float3 camToObj = cameraLookAtPoint - camera.pos;

	//camera.dir = normalize(camToObj);
	camera.yaw = -M_PI / 4.0f;
	camera.pitch = 0;
	camera.up  = { 0.0f, 1.0f, 0.0f };

	//camera.focal = camToObj.length();
	camera.focal = 5.0f;
	camera.aperture = 0.001f;

	camera.resolution = { (float)renderWidth, (float)renderHeight };
	camera.fov.x = 90.0f * Pi_over_180;

	if (g_settings->loadCameraAtInit)
	{
		LoadCameraFromFile(g_settings->inputCameraFileName);
	}

	camera.update();
}

void RayTracer::cleanup()
{
	// ---------------- Destroy surface objects ----------------------
	// triangle
	hipFree(triCountArray);
	hipFree(batchCountArray);

	hipFree(constTriangles);
	hipFree(triangles);

	// tlas
	hipFree(tlasAabbs);
	hipFree(tlasMorton);
	hipFree(tlasReorderIdx);
	hipFree(tlasBvhNodes);

	// bvh
	hipFree(morton);
	hipFree(reorderIdx);
	hipFree(bvhNodes);
	hipFree(aabbs);

	// color buffer
    hipDestroySurfaceObject(colorBufferA);
	hipDestroySurfaceObject(colorBufferB);
	hipFreeArray(colorBufferArrayA);
	hipFreeArray(colorBufferArrayB);

	// down sized color buffer
	hipDestroySurfaceObject(colorBuffer4);
	hipDestroySurfaceObject(colorBuffer16);
	hipDestroySurfaceObject(colorBuffer64);
	hipFreeArray(colorBufferArray4);
	hipFreeArray(colorBufferArray16);
	hipFreeArray(colorBufferArray64);

	// bloom buffer
	hipDestroySurfaceObject(bloomBuffer4);
	hipDestroySurfaceObject(bloomBuffer16);
	hipFreeArray(bloomBufferArray4);
	hipFreeArray(bloomBufferArray16);

	// normal depth buffer
	hipDestroySurfaceObject(normalDepthBufferA);
	hipDestroySurfaceObject(normalDepthBufferB);
	hipFreeArray(normalDepthBufferArrayA);
	hipFreeArray(normalDepthBufferArrayB);

	// motion vector buffer
	hipDestroySurfaceObject(motionVectorBuffer);
	hipFreeArray(motionVectorBufferArray);

	// sample count buffer
	hipDestroySurfaceObject(sampleCountBuffer);
	hipFreeArray(sampleCountBufferArray);

	// noise level buffer
	hipDestroySurfaceObject(noiseLevelBuffer);
	hipFreeArray(noiseLevelBufferArray);

	hipDestroySurfaceObject(colorBufferC);
	hipFreeArray(colorBufferArrayC);

	hipDestroySurfaceObject(bsdfOverPdfBuffer);
	hipFreeArray(bsdfOverPdfBufferArray);

	// ---------------------- destroy texture objects --------------------------
	hipDestroyTextureObject(sceneTextures.uv);
	hipFreeArray(texArrayUv);

	//hipDestroyTextureObject(sceneTextures.sandAlbedo);
	//hipDestroyTextureObject(sceneTextures.sandNormal);
	//if (texArraySandAlbedo != nullptr) hipFreeArray(texArraySandAlbedo);
	//if (texArraySandNormal != nullptr) hipFreeArray(texArraySandNormal);

	// sky
	//hipDestroyTextureObject(skyTex);
	hipDestroySurfaceObject(skyBuffer);
	hipFreeArray(skyArray);
	hipFree(skyCdf);

	// --------------------- free other gpu buffer ----------------------------
	// exposure and histogram
	hipFree(d_exposure);
	hipFree(d_histogram);

	// scene
	hipFree(d_spheres);
	hipFree(d_surfaceMaterials);
	hipFree(d_sceneAabbs);
	hipFree(d_materialsIdx);
	hipFree(d_sphereLights);

	hipFree(dumpFrameBuffer);

	// random
	h_randGen.clear();

	// free cpu buffer
	delete sceneAabbs;
	delete spheres;
	delete sphereLights;
}