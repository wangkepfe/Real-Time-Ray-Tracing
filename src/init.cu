#include "hip/hip_runtime.h"

#include "kernel.cuh"
#include "fileUtils.cuh"
#include "blueNoiseRandGenData.h"
#include "hip/hip_fp16.h"

template<typename T>
__global__ void InitBuffer(T val, SurfObj buffer, Int2 bufferSize)
{
	Int2 idx;
	idx.x = blockIdx.x * blockDim.x + threadIdx.x;
	idx.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx.x >= bufferSize.x || idx.y >= bufferSize.y) return;

	surf2Dwrite(val, buffer, idx.x, idx.y, hipBoundaryModeClamp);
}

void RayTracer::init(hipStream_t* cudaStreams)
{
	maxRenderWidth = 3840;
	maxRenderHeight = 2160;

	if (UseDynamicResolution)
	{
		renderWidth = maxRenderWidth;
		renderHeight = maxRenderHeight;
	}
	else
	{
		renderWidth = screenWidth;
		renderHeight = screenHeight;
	}

	uint i;

	// set streams
	streams = cudaStreams;

	// init cuda
	gpuDeviceInit(0);

	// scope for shorter cpu buffer lifetime
	{
		// load triangles
		std::vector<Triangle> h_triangles;
		//const chat* filename = "resources/models/testCube.obj";
		const char* filename = "resources/models/monkey.obj";
		LoadScene(filename, h_triangles);
		triCount = static_cast<uint>(h_triangles.size());

		GpuErrorCheck(hipMalloc((void**)& constTriangles, triCount * sizeof(Triangle)));
		GpuErrorCheck(hipMemcpy(constTriangles, h_triangles.data(), triCount * sizeof(Triangle), hipMemcpyHostToDevice));
	}

	// bvh
	GpuErrorCheck(hipMalloc((void**)& triangles, triCount * sizeof(Triangle)));
	GpuErrorCheck(hipMalloc((void**)& aabbs, triCount * sizeof(AABB)));

	GpuErrorCheck(hipMalloc((void**) &sceneBoundingBox, sizeof(AABB)));

	GpuErrorCheck(hipMalloc((void**)& morton, BVHcapacity * sizeof(uint)));
	GpuErrorCheck(hipMemset(morton, UINT_MAX, BVHcapacity * sizeof(uint)));

	GpuErrorCheck(hipMalloc((void**)& reorderIdx, BVHcapacity * sizeof(uint)));

	GpuErrorCheck(hipMalloc((void**)& bvhNodes, (triCount - 1) * sizeof(BVHNode)));
	GpuErrorCheck(hipMalloc((void**)& isAabbDone, (triCount - 1) * sizeof(uint)));
	GpuErrorCheck(hipMemset(isAabbDone, 0, (triCount - 1) * sizeof(uint)));

	// AABB
	int numAabbs = 2;
	sceneAabbs = new AABB[numAabbs];
	i = 0;
	sceneAabbs[i++] = AABB({0.0f, 0.0f, 0.0f}, 0.01f);
	sceneAabbs[i++] = AABB({0.0f, 0.0f, 0.0f}, 0.01f);

	// sphere
	numSpheres = 2;
	spheres    = new Sphere[numSpheres];
	i = 0;
	spheres[i++] = Sphere({0.0f, 1.0f, 4.0f}, 1.0f);
	spheres[i++] = Sphere({0.0f, 1.0f, -4.0f}, 1.0f);

	// surface materials
	const int numMaterials     = 10;
	SurfaceMaterial* materials = new SurfaceMaterial[numMaterials];
	i = 0;
	materials[i].type          = EMISSIVE;
	materials[i].albedo        = Float3(0.1f, 0.2f, 0.9f);
	++i;
	materials[i].type          = PERFECT_FRESNEL_REFLECTION_REFRACTION;
	++i;
	materials[i].type          = EMISSIVE;
	materials[i].albedo        = Float3(0.9f, 0.2f, 0.1f);
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].albedo        = Float3(0.9f);
	++i;
	materials[i].type          = MICROFACET_REFLECTION;
	materials[i].albedo        = Float3(0.9f);
	materials[i].F0            = Float3(0.56f, 0.57f, 0.58f);
	materials[i].alpha         = 0.05f;
	++i;
	materials[i].type          = MICROFACET_REFLECTION;
	materials[i].albedo        = Float3(0.9f);
	materials[i].F0            = Float3(0.56f, 0.57f, 0.58f);
	materials[i].alpha         = 0.01f;
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].useTex0       = true;
	materials[i].texId0        = 0;
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].albedo        = Float3(0.9f, 0.2f, 0.1f);
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].albedo        = Float3(0.2f, 0.9f, 0.1f);
	++i;
	materials[i].type          = LAMBERTIAN_DIFFUSE;
	materials[i].albedo        = Float3(0.1f, 0.2f, 0.9f);


	// number of objects
	const int numObjects = triCount + numSpheres;

	// material index
	int* materialsIdx = new int[numObjects];
	for (i = 0; i < triCount; ++i)
	{
		materialsIdx[i] = 1;
	}
	materialsIdx[i++] = 0;
	materialsIdx[i++] = 2;

	// light source
	numSphereLights = 2;
	sphereLights = new Sphere[numSphereLights];
	for (i = 0; i < numSphereLights; ++i)
	{
		sphereLights[i] = spheres[i];
	}

	// constant buffer
	cbo.frameNum = 0;
	cbo.bvhDebugLevel = -1;

	// launch param
	blockDim = dim3(8, 8, 1);
	gridDim = dim3(divRoundUp(renderWidth, blockDim.x), divRoundUp(renderHeight, blockDim.y), 1);

	scaleBlockDim = dim3(8, 8, 1);
	scaleGridDim = dim3(divRoundUp(screenWidth, scaleBlockDim.x), divRoundUp(screenHeight, scaleBlockDim.y), 1);

	// ------------------------ surface/texture object ---------------------------
	hipChannelFormatDesc format_color_RGB16_mask_A16 = cudaCreateChannelDescHalf4();
	hipChannelFormatDesc format_normal_R11_G10_B11_depth_R32 = hipCreateChannelDesc<float2>();
	hipChannelFormatDesc format_motionVector_UV16 = cudaCreateChannelDescHalf2();
	hipChannelFormatDesc format_sampleCount_R8 = hipCreateChannelDesc<uchar1>();
	hipChannelFormatDesc format_noiseLevel_R16 = cudaCreateChannelDescHalf1();

	// resource desription
	hipResourceDesc resDesc = {};
	resDesc.resType = hipResourceTypeArray;

	// texture description
	hipTextureDesc texDesc  = {};
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.addressMode[1]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	// array A: main render buffer
	GpuErrorCheck(hipMallocArray(&colorBufferArrayA, &format_color_RGB16_mask_A16, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArrayA;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBufferA, &resDesc));

	// array B: TAA buffer
	GpuErrorCheck(hipMallocArray(&colorBufferArrayB, &format_color_RGB16_mask_A16, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArrayB;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBufferB, &resDesc));

	// normal depth
	GpuErrorCheck(hipMallocArray(&normalDepthBufferArrayA, &format_normal_R11_G10_B11_depth_R32, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = normalDepthBufferArrayA;
	GpuErrorCheck(hipCreateSurfaceObject(&normalDepthBufferA, &resDesc));

	GpuErrorCheck(hipMallocArray(&normalDepthBufferArrayB, &format_normal_R11_G10_B11_depth_R32, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = normalDepthBufferArrayB;
	GpuErrorCheck(hipCreateSurfaceObject(&normalDepthBufferB, &resDesc));

	// motion vector buffer
	GpuErrorCheck(hipMallocArray(&motionVectorBufferArray, &format_motionVector_UV16, renderWidth, renderHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = motionVectorBufferArray;
	GpuErrorCheck(hipCreateSurfaceObject(&motionVectorBuffer, &resDesc));

	// sample count buffer
	GpuErrorCheck(hipMallocArray(&sampleCountBufferArray, &format_sampleCount_R8, gridDim.x, gridDim.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = sampleCountBufferArray;
	GpuErrorCheck(hipCreateSurfaceObject(&sampleCountBuffer, &resDesc));

	InitBuffer<<<dim3(divRoundUp(gridDim.x, 8), divRoundUp(gridDim.y, 8), 1), dim3(8, 8, 1)>>> (make_uchar1(1), sampleCountBuffer, Int2(gridDim.x, gridDim.y));

	// noise level uffer
	GpuErrorCheck(hipMallocArray(&noiseLevelBufferArray, &format_noiseLevel_R16, gridDim.x, gridDim.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = noiseLevelBufferArray;
	GpuErrorCheck(hipCreateSurfaceObject(&noiseLevelBuffer, &resDesc));

	InitBuffer<<<dim3(divRoundUp(gridDim.x, 8), divRoundUp(gridDim.y, 8), 1), dim3(8, 8, 1)>>> (make_ushort1(0), noiseLevelBuffer, Int2(gridDim.x, gridDim.y));

	// color buffer 1/4 size
	bufferSize4 = UInt2(divRoundUp(renderWidth, 4u), divRoundUp(renderHeight, 4u));
	gridDim4 = dim3(divRoundUp(bufferSize4.x, blockDim.x), divRoundUp(bufferSize4.y, blockDim.y), 1);
	GpuErrorCheck(hipMallocArray(&colorBufferArray4, &format_color_RGB16_mask_A16, bufferSize4.x, bufferSize4.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArray4;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBuffer4, &resDesc));

	// bloom buffer 1/4 size
	GpuErrorCheck(hipMallocArray(&bloomBufferArray4, &format_color_RGB16_mask_A16, bufferSize4.x, bufferSize4.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = bloomBufferArray4;
	GpuErrorCheck(hipCreateSurfaceObject(&bloomBuffer4, &resDesc));

	// color buffer 1/16 size
	bufferSize16 = UInt2(divRoundUp(bufferSize4.x, 4u), divRoundUp(bufferSize4.y, 4u));
	gridDim16 = dim3(divRoundUp(bufferSize16.x, blockDim.x), divRoundUp(bufferSize16.y, blockDim.y), 1);
	GpuErrorCheck(hipMallocArray(&colorBufferArray16, &format_color_RGB16_mask_A16, bufferSize16.x, bufferSize16.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArray16;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBuffer16, &resDesc));

	// bloom buffer 1/16 size
	GpuErrorCheck(hipMallocArray(&bloomBufferArray16, &format_color_RGB16_mask_A16, bufferSize16.x, bufferSize16.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = bloomBufferArray16;
	GpuErrorCheck(hipCreateSurfaceObject(&bloomBuffer16, &resDesc));

	// color buffer 1/64 size
	bufferSize64 = UInt2(divRoundUp(bufferSize16.x, 4u), divRoundUp(bufferSize16.y, 4u));
	gridDim64 = dim3(divRoundUp(bufferSize64.x, blockDim.x), divRoundUp(bufferSize64.y, blockDim.y), 1);
	GpuErrorCheck(hipMallocArray(&colorBufferArray64, &format_color_RGB16_mask_A16, bufferSize64.x, bufferSize64.y, hipArraySurfaceLoadStore));
	resDesc.res.array.array = colorBufferArray64;
	GpuErrorCheck(hipCreateSurfaceObject(&colorBuffer64, &resDesc));

	// ----------------------- sky buffer ------------------------
	GpuErrorCheck(hipMallocArray(&skyArray, &format_color_RGB16_mask_A16, skyWidth, skyHeight, hipArraySurfaceLoadStore));
	resDesc.res.array.array = skyArray;
	GpuErrorCheck(hipCreateSurfaceObject(&skyBuffer, &resDesc));
	GpuErrorCheck(hipCreateTextureObject(&skyTex, &resDesc, &texDesc, NULL));

	GpuErrorCheck(hipMalloc((void**)&skyCdf, skySize * sizeof(float)));
	GpuErrorCheck(hipMemset(skyCdf, 0, skySize * sizeof(float)));

	// ----------------------- GPU buffers -----------------------
	// exposure
	GpuErrorCheck(hipMalloc((void**)& d_exposure, 4 * sizeof(float)));
	float initExposureLum[4] = { 1.0f, 1.0f, 1.0f, 1.0f }; // (exposureValue, historyAverageLuminance, historyBrightThresholdLuminance, unused)
	GpuErrorCheck(hipMemcpy(d_exposure, initExposureLum, 4 * sizeof(float), hipMemcpyHostToDevice));

	// histogram
	GpuErrorCheck(hipMalloc((void**)& d_histogram, 64 * sizeof(uint)));

	// scene
	GpuErrorCheck(hipMalloc((void**)& d_spheres          , numSpheres *       sizeof(Sphere)));
	GpuErrorCheck(hipMalloc((void**)& d_sceneAabbs       , numAabbs *         sizeof(AABB)));
	GpuErrorCheck(hipMalloc((void**)& d_materialsIdx     , numObjects *       sizeof(int)));
	GpuErrorCheck(hipMalloc((void**)& d_surfaceMaterials , numMaterials *     sizeof(SurfaceMaterial)));
	GpuErrorCheck(hipMalloc((void**)& d_sphereLights     , numSphereLights *  sizeof(Float4)));

	GpuErrorCheck(hipMemcpy(d_spheres          , spheres      , numSpheres *      sizeof(Float4)         , hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(d_surfaceMaterials , materials    , numMaterials *    sizeof(SurfaceMaterial), hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(d_sceneAabbs       , sceneAabbs   , numAabbs *        sizeof(AABB)           , hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(d_materialsIdx     , materialsIdx , numObjects *      sizeof(int)            , hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(d_sphereLights     , sphereLights , numSphereLights * sizeof(Float4)         , hipMemcpyHostToDevice));

	// setup scene
	d_sceneGeometry.numSpheres      = numSpheres;
	d_sceneGeometry.spheres         = d_spheres;

	d_sceneGeometry.numAabbs        = numAabbs;
	d_sceneGeometry.aabbs           = d_sceneAabbs;

	d_sceneMaterial.numSphereLights = numSphereLights;
	d_sceneMaterial.sphereLights    = d_sphereLights;

	d_sceneMaterial.materials       = d_surfaceMaterials;
	d_sceneMaterial.materialsIdx    = d_materialsIdx;
	d_sceneMaterial.numMaterials    = numMaterials;

	d_sceneGeometry.triangles       = triangles;
	d_sceneGeometry.bvhNodes        = bvhNodes;
	d_sceneGeometry.numTriangles    = triCount;

	delete[] materials;
	delete[] materialsIdx;

	// cuda random
	h_randGen.init();
	d_randGen = h_randGen;

	// camera
	CameraSetup(cbo.camera);

	// textures
	texArrayUv = LoadTextureRgba8("resources/textures/colorChecker.png", sceneTextures.uv);
	//texArraySandAlbedo = LoadTextureRgb8("resources/textures/sand.png", sceneTextures.sandAlbedo);
	//texArraySandNormal = LoadTextureRgb8("resources/textures/sand_n.png", sceneTextures.sandNormal);

	// timer init
	timer.init();

	// set render dim to screen dim
	renderWidth = screenWidth;
	renderHeight = screenHeight;
}

void RayTracer::CameraSetup(Camera& camera)
{
	//cameraFocusPos = Float3(0, 1.0f, 0);
	//camera.pos = cameraFocusPos + Float3(7.3f, 2.0f, -6.9f);
	camera.pos = Float3(7.3f, 2.0f, -6.9f);

	//Float3 cameraLookAtPoint = cameraFocusPos;
	//Float3 camToObj = cameraLookAtPoint - camera.pos;

	//camera.dir = normalize(camToObj);
	camera.yaw = 0;
	camera.pitch = 0;
	camera.up  = { 0.0f, 1.0f, 0.0f };

	//camera.focal = camToObj.length();
	camera.focal = 5.0f;
	camera.aperture = 0.000001f;

	camera.resolution = { (float)renderWidth, (float)renderHeight };
	camera.fov.x = 90.0f * Pi_over_180;

	camera.update();
}

void RayTracer::cleanup()
{
	// ---------------- Destroy surface objects ----------------------
	// bvh
	hipFree(constTriangles);
	hipFree(triangles);
	hipFree(sceneAabbs);
	hipFree(sceneBoundingBox);
	hipFree(morton);
	hipFree(reorderIdx);
	hipFree(bvhNodes);
	hipFree(isAabbDone);

	// color buffer
    hipDestroySurfaceObject(colorBufferA);
	hipDestroySurfaceObject(colorBufferB);
	hipFreeArray(colorBufferArrayA);
	hipFreeArray(colorBufferArrayB);

	// down sized color buffer
	hipDestroySurfaceObject(colorBuffer4);
	hipDestroySurfaceObject(colorBuffer16);
	hipDestroySurfaceObject(colorBuffer64);
	hipFreeArray(colorBufferArray4);
	hipFreeArray(colorBufferArray16);
	hipFreeArray(colorBufferArray64);

	// bloom buffer
	hipDestroySurfaceObject(bloomBuffer4);
	hipDestroySurfaceObject(bloomBuffer16);
	hipFreeArray(bloomBufferArray4);
	hipFreeArray(bloomBufferArray16);

	// normal depth buffer
	hipDestroySurfaceObject(normalDepthBufferA);
	hipDestroySurfaceObject(normalDepthBufferB);
	hipFreeArray(normalDepthBufferArrayA);
	hipFreeArray(normalDepthBufferArrayB);

	// motion vector buffer
	hipDestroySurfaceObject(motionVectorBuffer);
	hipFreeArray(motionVectorBufferArray);

	// sample count buffer
	hipDestroySurfaceObject(sampleCountBuffer);
	hipFreeArray(sampleCountBufferArray);

	// noise level buffer
	hipDestroySurfaceObject(noiseLevelBuffer);
	hipFreeArray(noiseLevelBufferArray);

	// ---------------------- destroy texture objects --------------------------
	hipDestroyTextureObject(sceneTextures.sandAlbedo);
	hipDestroyTextureObject(sceneTextures.uv);
	hipDestroyTextureObject(sceneTextures.sandNormal);
	if (texArraySandAlbedo != nullptr) hipFreeArray(texArraySandAlbedo);
	if (texArrayUv         != nullptr) hipFreeArray(texArrayUv);
	if (texArraySandNormal != nullptr) hipFreeArray(texArraySandNormal);

	// sky
	hipDestroyTextureObject(skyTex);
	hipDestroySurfaceObject(skyBuffer);
	hipFreeArray(skyArray);
	hipFree(skyCdf);

	// --------------------- free other gpu buffer ----------------------------
	// exposure and histogram
	hipFree(d_exposure);
	hipFree(d_histogram);

	// scene
	hipFree(d_spheres);
	hipFree(d_surfaceMaterials);
	hipFree(d_sceneAabbs);
	hipFree(d_materialsIdx);
	hipFree(d_sphereLights);

	// random
	h_randGen.clear();

	// free cpu buffer
	delete sceneAabbs;
	delete spheres;
	delete sphereLights;
}